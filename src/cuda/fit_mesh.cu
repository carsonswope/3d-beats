#include "hip/hip_runtime.h"
#include <cu_utils.hpp>

extern "C" {__global__
void calc_image_cost(
        int dim_x,
        int dim_y,
        uint16* _d0,
        uint16* _d1,
        uint16* _labels, // for d0 evaluated
        uint16 target_label,
        float* cost) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dim_x || y >= dim_y) return;

    Array2d<uint16> d0(_d0, {dim_y, dim_x});
    Array2d<uint16> d1(_d1, {dim_y, dim_x});
    Array2d<uint16> labels(_labels, {dim_y, dim_x});

    const auto d0_val = d0.get({y, x});
    const auto d1_val = d1.get({y, x});
    const auto label_val = labels.get({y, x});

    // Various cost conditions here!

    // First case: original image did not have a value for this pixel.
    // Rendered image is allowed to have a pixel here at no cost
    if (d0_val == 0) {
        return;
    }

    const float BOUNDARY_MISMATCH_COST = 1000.;

    if (label_val == target_label && d1_val == 0) {
        atomicAdd(cost, BOUNDARY_MISMATCH_COST);
        return;
    }

    if (label_val != target_label && d1_val != 0) {
        if (d1_val > d0_val) {
            atomicAdd(cost, BOUNDARY_MISMATCH_COST);
        }
        return;
    }

    if (label_val == target_label && d1_val != 0) {
        const float diff = abs((d0_val * 1.f) - (d1_val * 1.f));
        atomicAdd(cost, 0.01 * diff*diff);        
        return;
    }

    // if (label_val == target_label) {
    //     if (d1_val == 0) {
    //         // expected there to be a pixel here, but rendered image did not have one!
    //         atomicAdd(cost, BOUNDARY_MISMATCH_COST);
    //     } else {
    //         // pixel here is expected. compare depth values!
    //         const float diff = abs((d0_val * 1.f) - (d1_val * 1.f));
    //         atomicAdd(cost, diff);
    //     }
    // } else {
    //     // label does not match
    //     if (d1_val == 0) {
    //         // good, label doesn't match so we don't expect rendered image to have written here
    //     } else {
    //         // bad - we wrote a pixel here that was not expected.
    //         if (d1_val > d0_val) {
    //             // if rendered pixel has depth greater than original depth,
    //             // then no cost, because it would have failed the depth test
    //         } else {
    //             atomicAdd(cost, BOUNDARY_MISMATCH_COST);
    //         }
    //     }
    // }

}}
