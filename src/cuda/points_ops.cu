#include "hip/hip_runtime.h"
#include <cu_utils.hpp>

// given a depth image, 
extern "C" {__global__
void deproject_points(
        int4 imgs_dim, // (num_images, dimx, dimy)
        float2 pp, // (ppx, ppy)
        float f, // focal length
        uint16* _imgs,
        float4* _pts) {
        
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;
    const int y = blockIdx.z * blockDim.z + threadIdx.z;

    const int num_images = imgs_dim.x;
    const int2 img_dim = {imgs_dim.y, imgs_dim.z};

    if (i >= num_images || x >= img_dim.x || y >= img_dim.y) return;

    Array3d<uint16> imgs(_imgs, {num_images,img_dim.y,img_dim.x});
    Array3d<float4> pts(_pts, {num_images,img_dim.y,img_dim.x}, {0., 0., 0., 0.});
    
    const uint16 d = imgs.get({i, y, x});
    if (d > 0) {
        const float d_ = d * 1.f;
        const float4 p{
            d_ * (x - pp.x) / f,
            d_ * (y - pp.y) / f,
            d_,
            1.,
        };

        pts.set({i, y, x}, p);
    }
}}

// given a depth image, 
extern "C" {__global__
void depths_from_points(
        int4 imgs_dim, // (num_images, dimx, dimy)
        // float2 pp, // (ppx, ppy)
        // float f, // focal length
        uint16* _imgs,
        float4* _pts) {
        
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;
    const int y = blockIdx.z * blockDim.z + threadIdx.z;

    const int num_images = imgs_dim.x;
    const int2 img_dim = {imgs_dim.y, imgs_dim.z};

    if (i >= num_images || x >= img_dim.x || y >= img_dim.y) return;

    Array3d<uint16> imgs(_imgs, {num_images,img_dim.y,img_dim.x});
    Array3d<float4> pts(_pts, {num_images,img_dim.y,img_dim.x}, {0., 0., 0., 0.});
    
    const float4 pos = pts.get({i, y, x});
    if (pos.w > 0.f) {
        imgs.set({i, y, x}, (uint16)pos.z);
    }
}}

extern "C" {__global__
void transform_points(int num_pts, glm::vec4* pts, glm::mat4 t) {
    
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= num_pts) return;

    glm::vec4 p = pts[i];
    if (p.w != 1.) return;
    auto new_p = glm::transpose(t) * p;
    pts[i] = new_p;

}}

extern "C" {__global__
void setup_depth_image_for_forest(
        int NUM_PIXELS,
        glm::vec4* pts,
        uint16* depth) {

    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= NUM_PIXELS) return;

    const uint16 d = depth[i];
    const glm::vec4 p = pts[i];

    if (d == 0 || p.w == 0) {
        depth[i] = 65535;
    }

}}

extern "C" {__global__
void apply_point_mapping(
        int IMG_DIM_X,
        int IMG_DIM_Y,
        int NUM_COLORS,
        uint8* _colors,
        uint8* _color_image) {
    
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= IMG_DIM_X || y >= IMG_DIM_Y) return;

    Array2d<uint8> colors(_colors, {NUM_COLORS, 3});
    Array3d<uint8> color_image(_color_image, {IMG_DIM_Y, IMG_DIM_X, 3});

    auto* color_image_pixel = color_image.get_ptr({y, x, 0});
    if (color_image_pixel[0] + color_image_pixel[1] + color_image_pixel[2] == 0) return;

    float best_squared_diff = -1.f;
    uint8* best_colors_ptr = nullptr;

    for (int i = 0; i < NUM_COLORS; i++) {
        auto* test_color = colors.get_ptr({i, 0});
        float squared_diff = 0;
        for (int j = 0; j < 3; j++) {
            const float diff = (color_image_pixel[j] * 1.f) - test_color[j];
            squared_diff += diff * diff;
        }
        if (best_colors_ptr == nullptr || squared_diff < best_squared_diff) {
            best_squared_diff = squared_diff;
            best_colors_ptr = test_color;
        }
    }

    // or memcpy..
    for (int j =0; j < 3; j++) {
        color_image_pixel[j] = best_colors_ptr[j];
    }
}}

extern "C" {__global__
void split_pixels_by_nearest_color(
        int IMG_DIM_X,
        int IMG_DIM_Y,
        int NUM_COLORS,
        uint8* _colors,
        uint8* _color_image,
        uint64* _pixel_counts_per_group) {
    
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= IMG_DIM_X || y >= IMG_DIM_Y) return;

    Array2d<uint8> colors(_colors, {NUM_COLORS, 3});
    Array3d<uint8> color_image(_color_image, {IMG_DIM_Y, IMG_DIM_X, 3});
    Array2d<uint64> pixel_counts_per_group(_pixel_counts_per_group, {NUM_COLORS, 5}); // (num_pixels, sum_r, sum_g, sum_b, sum_cost)
    
    float best_squared_diff = -1.f;
    int best_colors_idx = -1;

    auto* color_image_pixel = color_image.get_ptr({y, x, 0});
    if (color_image_pixel[0] + color_image_pixel[1] + color_image_pixel[2] == 0) return;

    for (int i = 0; i < NUM_COLORS; i++) {
        auto* test_color = colors.get_ptr({i, 0});
        float squared_diff = 0;
        for (int j = 0; j < 3; j++) {
            const float diff = (color_image_pixel[j] * 1.f) - test_color[j];
            squared_diff += diff * diff;
        }
        if (best_colors_idx == -1 || squared_diff < best_squared_diff) {
            best_squared_diff = squared_diff;
            best_colors_idx = i;
        }
    }


    uint64* p = pixel_counts_per_group.get_ptr({best_colors_idx, 0});
    atomicAdd(p + 0, 1);
    atomicAdd(p + 1, uint64(color_image_pixel[0]));
    atomicAdd(p + 2, uint64(color_image_pixel[1]));
    atomicAdd(p + 3, uint64(color_image_pixel[2]));
    atomicAdd((double*)p+4, (double)best_squared_diff);

    // or memcpy..
    // for (int j =0; j < 3; j++) {
        // color_image_pixel[j] = best_colors_ptr[j];
    // }
}}


extern "C" {__global__
void make_rgba_from_labels(
        int IMG_DIM_X,
        int IMG_DIM_Y,
        int NUM_COLORS,
        uint16* _labels,
        uint8* _colors,
        uint8* _color_image) {
    
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= IMG_DIM_X || y >= IMG_DIM_Y) return;

    Array2d<uint16> labels(_labels, {IMG_DIM_Y, IMG_DIM_X});
    Array2d<uint8> colors(_colors, {NUM_COLORS, 4});
    Array3d<uint8> color_image(_color_image, {IMG_DIM_Y, IMG_DIM_X, 4});
    
    const auto l = labels.get({y, x});
    if (l == 0 || l == MAX_UINT16) return;

    auto* color_img_ptr = color_image.get_ptr({y, x, 0});
    auto* color_ptr = colors.get_ptr({l - 1, 0});
    memcpy(color_img_ptr, color_ptr, sizeof(uint8) * 4); // should evaluate to just 4 bytes..
}}
