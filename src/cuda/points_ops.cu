#include "hip/hip_runtime.h"
#include <cu_utils.hpp>

// given a depth image, 
extern "C" {__global__
void deproject_points(
        int4 imgs_dim, // (num_images, dimx, dimy)
        float2 pp, // (ppx, ppy)
        float f, // focal length
        uint16* _imgs,
        float4* _pts) {
        
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;
    const int y = blockIdx.z * blockDim.z + threadIdx.z;

    const int num_images = imgs_dim.x;
    const int2 img_dim = {imgs_dim.y, imgs_dim.z};

    if (i >= num_images || x >= img_dim.x || y >= img_dim.y) return;

    Array3d<uint16> imgs(_imgs, {num_images,img_dim.y,img_dim.x});
    Array3d<float4> pts(_pts, {num_images,img_dim.y,img_dim.x}, {0., 0., 0., 0.});
    
    const uint16 d = imgs.get({i, y, x});
    if (d > 0) {
        const float d_ = d * 1.f;
        const float4 p{
            d_ * (x - pp.x) / f,
            d_ * (y - pp.y) / f,
            d_,
            1.,
        };

        pts.set({i, y, x}, p);
    }
}}

// given a depth image, 
extern "C" {__global__
void depths_from_points(
        int4 imgs_dim, // (num_images, dimx, dimy)
        // float2 pp, // (ppx, ppy)
        // float f, // focal length
        uint16* _imgs,
        float4* _pts) {
        
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;
    const int y = blockIdx.z * blockDim.z + threadIdx.z;

    const int num_images = imgs_dim.x;
    const int2 img_dim = {imgs_dim.y, imgs_dim.z};

    if (i >= num_images || x >= img_dim.x || y >= img_dim.y) return;

    Array3d<uint16> imgs(_imgs, {num_images,img_dim.y,img_dim.x});
    Array3d<float4> pts(_pts, {num_images,img_dim.y,img_dim.x}, {0., 0., 0., 0.});
    
    const float4 pos = pts.get({i, y, x});
    if (pos.w > 0.f) {
        imgs.set({i, y, x}, (uint16)pos.z);
    }
}}

extern "C" {__global__
void transform_points(int num_pts, glm::vec4* pts, glm::mat4 t) {
    
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= num_pts) return;

    glm::vec4 p = pts[i];
    if (p.w != 1.) return;
    auto new_p = glm::transpose(t) * p;
    pts[i] = new_p;

}}

extern "C" {__global__
void find_plane_ransac(
        int NUM_RANDOM_GUESSES,
        float PLANE_Z_OUTLIER_THRESHOLD,
        int NUM_PTS,
        glm::vec4* pts,
        glm::mat4* candidate_planes,
        int* num_inliers) {
    
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= NUM_PTS) return;

    const auto pt = pts[i];
    if (pt.w != 1.) return;

    for (int j = 0; j < NUM_RANDOM_GUESSES; j++) {

        const auto t = candidate_planes[j];
        const auto new_pt = glm::transpose(t) * pt;
        if (new_pt.z < PLANE_Z_OUTLIER_THRESHOLD && new_pt.z > -PLANE_Z_OUTLIER_THRESHOLD) {
            atomicAdd(num_inliers + j, 1);
        }
    }
}}


extern "C" {__global__
void filter_points_by_plane(
        int NUM_PTS,
        float PLANE_Z_FILTER_THRESHOLD,
        glm::vec4* pts){

    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= NUM_PTS) return;

    const auto pt = pts[i];
    if (pt.w != 1.) return;

    if (pt.z > -PLANE_Z_FILTER_THRESHOLD) {
        pts[i] = {0., 0., 0., 0.};
        // .x = 0.;
        // pts[i].y = 0.;
        // pts[i].z = 0.;
        // pts[i].w = 0.;
    }
    
}}


extern "C" {__global__
void make_plane_candidates(
        int NUM_CANDIDATES,
        int IMG_DIM_X,
        int IMG_DIM_Y,
        float* _rand,
        glm::vec4* pts,
        glm::mat4* plane_candidates){

    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= NUM_CANDIDATES) return;

    Array2d<float> rand_arr(_rand, {NUM_CANDIDATES, 32});
    
    glm::vec4 plane_pts[3];
    int plane_pts_set = 0;
    int rand_j = 0;

    while (plane_pts_set < 3 && rand_j < 32) {
        int r = __float2int_rd(rand_arr.get({i, rand_j}) * IMG_DIM_X * IMG_DIM_Y);
        glm::vec4 p = pts[r];
        if (p.z > 0.) {
            plane_pts[plane_pts_set++] = p;
        }
        rand_j++;
    }

    glm::vec3 v0 = glm::normalize((plane_pts[1] - plane_pts[0]).xyz());
    glm::vec3 v1 = glm::normalize((plane_pts[2] - plane_pts[0]).xyz());

    glm::vec3 z_axis = glm::normalize(glm::cross(v0, v1));
    glm::vec3 x_axis = v0;
    glm::vec3 y_axis = glm::normalize(glm::cross(z_axis, x_axis));

    glm::mat4 tf_mat = glm::mat4(1.f);
    tf_mat[0] = glm::vec4{x_axis, 0.f};
    tf_mat[1] = glm::vec4{y_axis, 0.f};
    tf_mat[2] = glm::vec4{z_axis, 0.f};
    tf_mat[3] = glm::vec4{-plane_pts[0].xyz(), 1.f};

    plane_candidates[i] = glm::transpose(tf_mat);

}}


extern "C" {__global__
void setup_depth_image_for_forest(
        int NUM_PIXELS,
        glm::vec4* pts,
        uint16* depth) {

    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= NUM_PIXELS) return;

    const uint16 d = depth[i];
    const glm::vec4 p = pts[i];

    if (d == 0 || p.w == 0) {
        depth[i] = 65535;
    }

}}

extern "C" {__global__
void apply_point_mapping(
        int IMG_DIM_X,
        int IMG_DIM_Y,
        int NUM_COLORS,
        uint8* _colors,
        uint8* _color_image) {
    
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= IMG_DIM_X || y >= IMG_DIM_Y) return;

    Array2d<uint8> colors(_colors, {NUM_COLORS, 3});
    Array3d<uint8> color_image(_color_image, {IMG_DIM_Y, IMG_DIM_X, 3});

    auto* color_image_pixel = color_image.get_ptr({y, x, 0});
    if (color_image_pixel[0] + color_image_pixel[1] + color_image_pixel[2] == 0) return;

    float best_squared_diff = -1.f;
    uint8* best_colors_ptr = nullptr;

    for (int i = 0; i < NUM_COLORS; i++) {
        auto* test_color = colors.get_ptr({i, 0});
        float squared_diff = 0;
        for (int j = 0; j < 3; j++) {
            const float diff = (color_image_pixel[j] * 1.f) - test_color[j];
            squared_diff += diff * diff;
        }
        if (best_colors_ptr == nullptr || squared_diff < best_squared_diff) {
            best_squared_diff = squared_diff;
            best_colors_ptr = test_color;
        }
    }

    // or memcpy..
    for (int j =0; j < 3; j++) {
        color_image_pixel[j] = best_colors_ptr[j];
    }
}}

extern "C" {__global__
void split_pixels_by_nearest_color(
        int IMG_DIM_X,
        int IMG_DIM_Y,
        int NUM_COLORS,
        uint8* _colors,
        uint8* _color_image,
        // int* _pixels_per_group,
        uint64* _pixel_counts_per_group) {
    
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= IMG_DIM_X || y >= IMG_DIM_Y) return;

    Array2d<uint8> colors(_colors, {NUM_COLORS, 3});
    Array3d<uint8> color_image(_color_image, {IMG_DIM_Y, IMG_DIM_X, 3});
    Array2d<uint64> pixel_counts_per_group(_pixel_counts_per_group, {NUM_COLORS, 5}); // (num_pixels, sum_r, sum_g, sum_b, sum_cost)
    
    float best_squared_diff = -1.f;
    int best_colors_idx = -1;

    auto* color_image_pixel = color_image.get_ptr({y, x, 0});
    if (color_image_pixel[0] + color_image_pixel[1] + color_image_pixel[2] == 0) return;

    for (int i = 0; i < NUM_COLORS; i++) {
        auto* test_color = colors.get_ptr({i, 0});
        float squared_diff = 0;
        for (int j = 0; j < 3; j++) {
            const float diff = (color_image_pixel[j] * 1.f) - test_color[j];
            squared_diff += diff * diff;
        }
        // if (true) {
        if (best_colors_idx == -1 || squared_diff < best_squared_diff) {
            best_squared_diff = squared_diff;
            best_colors_idx = i;
        }
    }

    // if (best_colors_idx == -1 || best_colors_idx >= NUM_COLORS) {
    //     printf("err! %i %i\n", x, y);
    // }

    // if (x == 0 && y ==0 ){ 
    //     printf("nc: %i\n", NUM_COLORS);
    //     printf("best_colrosidx: %i\n", best_colors_idx);
    //     printf("best sq df: %f\n", best_squared_diff);
    // }
    // if (best_colors_idx != 0) {
        // printf("huh?\n");
    // }

    uint64* p = pixel_counts_per_group.get_ptr({best_colors_idx, 0});
    atomicAdd(p + 0, 1);
    atomicAdd(p + 1, uint64(color_image_pixel[0]));
    atomicAdd(p + 2, uint64(color_image_pixel[1]));
    atomicAdd(p + 3, uint64(color_image_pixel[2]));
    atomicAdd((double*)p+4, (double)best_squared_diff);


    // auto pixel_num = atomicAdd(pixel_counts_per_group, 1);

    // or memcpy..
    // for (int j =0; j < 3; j++) {
        // color_image_pixel[j] = best_colors_ptr[j];
    // }
}}
  



// setup_depth_image_for_forest